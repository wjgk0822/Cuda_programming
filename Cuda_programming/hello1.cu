#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void hello_from_gpu()
{
	printf("Hello world from the GPU!\n");
}

int main(void)
{
	hello_from_gpu<<<1, 1>>>();
	hipDeviceSynchronize();
	return 0;
}