#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void hello_from_gpu()
{
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	printf("Hello world from block %d and thread %d!\n", bid, tid);

}

int main(void)
{
	hello_from_gpu << <2, 4 >> > ();
	hipDeviceSynchronize();
	return 0;
}