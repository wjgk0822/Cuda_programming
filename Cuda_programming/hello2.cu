#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void hello_from_gpu()
{
	printf("Hello world from the GPU!\n");
}

int main(void)
{
	hello_from_gpu << <2, 4 >> > ();
	hipDeviceSynchronize();
	return 0;
}